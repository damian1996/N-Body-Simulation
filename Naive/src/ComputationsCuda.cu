#include "hip/hip_runtime.h"
#include "ComputationsCuda.h"

// const double G = 6.674*(1e-11);
const double x = 1.0 / (3.086 * 1e13);   //[km -> pc]
const double G = 4.3 * (1e-3) / (x * x); //[pc^3/Mo*s^2]
const double EPS = 0.01f;

template <typename T>
__global__ void NaiveSim(T *pos, T *velo, T *weigh, int N, double dt) {
  // printf("%f\n", pos->arr[thid]);
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  float posx = pos[thid * 3], posy = pos[thid * 3 + 1],
        posz = pos[thid * 3 + 2], weighI = weigh[thid];
  float forcex = 0.0f, forcey = 0.0f, forcez = 0.0f;
  for (int j = 0; j < N; j++) {
    if (j != thid) {
      float distX = pos[j * 3] - posx;
      float distY = pos[j * 3 + 1] - posy;
      float distZ = pos[j * 3 + 2] - posz;
      float dist = (distX * distX + distY * distY + distZ * distZ) + EPS * EPS;
      if (fabs(distX) > 1e-10 && fabs(distY) > 1e-10 && fabs(distZ) > 1e-10) {
        float F = G * (weighI * weigh[j]);
        forcex += F * distX / dist;
        forcey += F * distY / dist;
        forcez += F * distZ / dist;

        /*if(dist>=EPS) {
            forcex += F*distX/dist;
            forcey += F*distY/dist;
        } else {
          forcex += F*distX;
          forcey += F*distY;
        }*/
      }
    }
  }
  double acc = forcex / weighI; // pc / s2
  pos[thid * 3] += velo[thid * 3] * dt + acc * dt * dt / 2;
  velo[thid * 3] += acc * dt;

  acc = forcey / weighI;
  pos[thid * 3 + 1] += velo[thid * 3 + 1] * dt + acc * dt * dt / 2;
  velo[thid * 3 + 1] += acc * dt;

  acc = forcez / weighI;
  pos[thid * 3 + 2] += velo[thid * 3 + 2] * dt + acc * dt * dt / 2;
  velo[thid * 3 + 2] += acc * dt;

  __syncthreads();
}

void Computations::NaiveSimBridgeThrust(type &pos, int N, double dt) {
  // type = thrust::device_vector<double>
  thrust::device_vector<double> posD = pos;

  double *d_positions = thrust::raw_pointer_cast(posD.data());
  double *d_velocities = thrust::raw_pointer_cast(veloD.data());
  double *d_weights = thrust::raw_pointer_cast(weightsD.data());

  float zzpx = 0.0f, zzpy = 0.0f, zzpz = 0.0f;
  for (int i = 0; i < N; i++) {
    zzpx += (weightsD[i] * veloD[i * 3]);
    zzpy += (weightsD[i] * veloD[i * 3 + 1]);
    zzpz += (weightsD[i] * veloD[i * 3 + 2]);
  }
  // std::cout << "Pedy : " << zzpx << "  " << zzpy << std::endl;

  NaiveSim<<<64, (N + 63) / 64>>>(d_positions, d_velocities, d_weights, N, dt);

  pos = posD;
  // thrust::copy(weightsD.begin(), weightsD.end(),
  // std::ostream_iterator<double>(std::cout, " "));
}

// https://www.bu.edu/pasi/files/2011/07/Lecture6.pdf
// https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html
// https://stackoverflow.com/questions/4176762/passing-structs-to-cuda-kernels
// https://codeyarns.com/2011/02/16/cuda-dim3/
// http://developer.download.nvidia.com/CUDA/training/introductiontothrust.pdf
// https://groups.google.com/forum/#!topic/thrust-users/4EaWLGeJOO8
// https://github.com/thrust/thrust/blob/master/examples/cuda/unwrap_pointer.cu
